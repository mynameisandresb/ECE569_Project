#include "hip/hip_runtime.h"
#include "utils.h"

/*
* Specify the size of a THREAD BLOCK of sizze THREAD_SIZE x THREAD_SIZE
*/
#define THREAD_SIZE 11
/*
* Flag whether to use separable gaussian filter or 2D
*/
#define SEPARATED_GAUSSIAN_FILTER 1

/**
* CUDA Kernel for DSGM
*/
__global__
void gaussian_background_kernel(unsigned char * const d_frame,
                            unsigned char* const d_amean, 
                            unsigned char* const d_cmean,
                            unsigned char* const d_avar,
                            unsigned char* const d_cvar,
                            unsigned char* const d_bin,
                            int * const d_aage,
                            int * const d_cage,
                       int numRows, int numCols)
{
  const size_t r = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t c = blockIdx.y * blockDim.y + threadIdx.y;
  const size_t index = r * numCols + c;

  // Make sure you're in memory bounds
  if (index < numRows * numCols)
  {

    float alpha, V;
    int adiff;
    int cdiff;

    float pixel = d_frame[index];
    float ameanpixel = d_amean[index];
    float avarpixel = d_avar[index];
    float cmeanpixel = d_cmean[index];
    float cvarpixel = d_cvar[index];

    adiff = pixel - ameanpixel;
    cdiff = pixel - cmeanpixel;
    // If within some threshold of the absolute background, update
    if(adiff*adiff < 9 * avarpixel){
        alpha = 1.0f / (float)d_aage[index];
        d_amean[index] = (1.0f-alpha) * ameanpixel + (alpha) * pixel;
        adiff = d_amean[index] - pixel;
        V = adiff*adiff;
        d_avar[index] = (1.0f-alpha) * avarpixel + alpha * V;
        d_aage[index]++;
    }
    //otherwise if in some threshold of the candidate, update
    else if(cdiff*cdiff < 9 * cvarpixel){
        alpha = 1.0f / (float)d_cage[index];
        d_cmean[index] = (1.0f-alpha) * cmeanpixel + (alpha) * pixel;
        cdiff = d_cmean[index] - pixel;
        V = cdiff*cdiff;
        d_cvar[index] = (1.0f-alpha) * cvarpixel + alpha * V;
        d_cage[index]++;
    }
    //otherwise reset candidate
    else{      
        d_cmean[index] = pixel;
        d_cvar[index] = 255;
        d_cage[index] = 1;
    }

    //if candidate age is larger
    if(d_cage[index] > d_aage[index]){
      //swap the candidate to the absolute
      d_amean[index] = d_cmean[index];
      d_avar[index] = d_cvar[index];
      d_aage[index] = d_cage[index];

      //reset the candidate model
      d_cmean[index] = pixel;
      d_cvar[index] = 255;
      d_cage[index] = 1;
    }

    adiff = pixel - d_amean[index];

    // Update motion mask
    if (adiff*adiff <= 60) {
        //background
        d_bin[index]= 0;
    } else {
        //foreground
        d_bin[index] = 255;
    }
  }
}

/**
* A call to the CUDA kernel, specify the block size first
*/
void gaussian_background(unsigned char* const d_frame,
                            unsigned char* const d_amean, 
                            unsigned char* const d_cmean,
                            unsigned char* const d_avar,
                            unsigned char* const d_cvar,
                            unsigned char* const d_bin,
                            int * const d_aage,
                            int * const d_cage,
                            size_t numRows, size_t numCols)
{
  const dim3 blockSize(THREAD_SIZE, THREAD_SIZE, 1);
  const dim3 gridSize(numRows / THREAD_SIZE + 1, numCols / THREAD_SIZE + 1, 1); 
  gaussian_background_kernel<<<gridSize, blockSize>>>(d_frame, d_amean, d_cmean, 
                                              d_avar, d_cvar, d_bin, d_aage, d_cage,
                                                numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}

/**
* 2D, non-separable Gaussian filter kernel
*/
__global__
void gaussian_filter_kernel(unsigned char* d_frame,
                     unsigned char* d_blurred,
                     const float* const d_gfilter,
                     size_t d_filter_width,
                     size_t d_filter_height,
                     size_t numRows, size_t numCols){

  const size_t r = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t c = blockIdx.y * blockDim.y + threadIdx.y;
  const size_t index = r * numCols + c; //the center pixel being blurred

  if (index >= numRows * numCols) return;

  int halfway_point = d_filter_width/2;
  float blurred_pixel = 0.0f;

  // Iterate over 2D Gaussian kernel
  for (int i = -halfway_point; i <= halfway_point; ++i){ 
    for (int j = -halfway_point; j <= halfway_point; ++j){ 
            // get the location of the desired pixel, clamped to borders of the image
            int h = fmin(fmax((float)(r + i), 0.f), (float)(numRows-1)); 
            int w = fmin(fmax((float)(c + j), 0.f), (float)(numCols-1)); 
            int current_pixel_id = w + numCols * h;
            float current_pixel = static_cast<float>(d_frame[current_pixel_id]); 

            // now, get the associated weight in the filter
            current_pixel_id = (i + halfway_point) * d_filter_width + j + halfway_point; 
            float weight = d_gfilter[current_pixel_id]; 
            blurred_pixel += current_pixel * weight; 
        } 
    } 
 
  d_blurred[index] = static_cast<int>(blurred_pixel); 
}

/**
* Median filter CUDA kernel
* NOTE: parts of this were taken from: http://stackoverflow.com/questions/19634328/2d-cuda-median-filter-optimization
*/
__global__
void median_filter_kernel(unsigned char* d_frame,
                     unsigned char* d_blurred,
                     size_t numRows, size_t numCols){

    const int size = 9;
    unsigned short surround[9];

    int iterator, i;

    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int index = x * numCols + y;   

    // if out of bounds return
    if( (x >= (numRows)) || (y >= numCols) || (x < 0) || (y < 0)) return;

    //if border, don't blur
    if( (x == (numRows - 1)) || (y == numCols - 1) || (x == 0) || (y == 0)){
      d_blurred[index] = d_frame[index];
    }

    // get the surrounding pixels and fill a local array
    iterator = 0;
    for (int r = x - 1; r <= x + 1; r++) {
        for (int c = y - 1; c <= y + 1; c++) {
            surround[iterator] = d_frame[r*numCols+c];
            iterator++;
        }
    }


    // simple sorting
    int middle = (size/2)+1;
    for (i=0; i<=middle; i++) {
        int minval=i;
        for (int l=i+1; l<size; l++){
          if (surround[l] < surround[minval]){
             minval=l;
          }
        } 
        unsigned short temp = surround[i];
        surround[i]=surround[minval];
        surround[minval]=temp;
    }

    // Set to the median value
    d_blurred[index] = surround[middle]; 
}

/**
* Call to the gaussian filter CUDA kernel to specify the blocksize
*/
void gaussian_filter(unsigned char* d_frame,
                     unsigned char* d_blurred,
                     const float* const d_gfilter,
                     size_t d_filter_width,
                     size_t d_filter_height,
                     size_t numRows, size_t numCols)
{

  const dim3 blockSize(THREAD_SIZE, THREAD_SIZE, 1);
  const dim3 gridSize(numRows / THREAD_SIZE + 1, numCols / THREAD_SIZE + 1, 1); 
  gaussian_filter_kernel<<<gridSize, blockSize>>>(d_frame, d_blurred, d_gfilter, 
                                                  d_filter_width, d_filter_height, 
                                                  numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

/**
* A separable gaussian filter kernel
*/
__global__
void gaussian_filter_kernel_separable(unsigned char* d_frame,
                     unsigned char* d_blurred,
                     const float* const d_gfilter,
                     size_t d_filter_size,
                     size_t numRows, size_t numCols, bool x_direction){

  const int r = blockIdx.x * blockDim.x + threadIdx.x;
  const int c = blockIdx.y * blockDim.y + threadIdx.y;
  const int index = r * numCols + c; //the center pixel being blurred

  // bounds check
  if ((r >= numRows) || (c >= numCols))
  {
    return;
  }

  int halfway_point = d_filter_size/2;
  unsigned char blurred_pixel = 0;
  int h, w, temp;
  
  //iterate over 1 dimensional gaussian kernel for convolution
  for (int j = -halfway_point; j <= halfway_point; ++j){ 
    //get the desired direction and clamp to borders    
    if(x_direction){
      temp = r+j;
      if(temp > numRows-1) temp = numRows-1;
      else if(temp < 0) temp = 0;
      h = temp; 
      w = c;
    }
    else{
      temp = c+j;
      if(temp > numCols-1) temp = numCols-1;
      else if(temp < 0) temp = 0;

      w = temp; 
      h = r;
    }
    
    size_t current_pixel_id = w + numCols * h;
    float current_pixel = d_frame[current_pixel_id]; 

    // now, get the associated weight in the filter
    current_pixel_id = (j + halfway_point); 
    float weight = d_gfilter[current_pixel_id]; 
    unsigned char t = current_pixel * weight; 
    blurred_pixel += t;
  } 


  d_blurred[index] = blurred_pixel; 
}

/**
* Call to the separable filter
*/
void gaussian_filter_separable(unsigned char* d_frame,
                     unsigned char* d_blurred,
                     unsigned char* d_blurred_temp,
                     const float* const d_gfilter,
                     size_t d_filter_size,
                     size_t numRows, size_t numCols)
{

  const dim3 blockSize(THREAD_SIZE, THREAD_SIZE, 1);
  const dim3 gridSize(numRows / THREAD_SIZE + 1, numCols / THREAD_SIZE + 1, 1); 
  // once in the x direction
  gaussian_filter_kernel_separable<<<gridSize, blockSize>>>(d_frame, d_blurred_temp, d_gfilter, 
                                                  d_filter_size, 
                                                  numRows, numCols, true);
  //once in the y
  gaussian_filter_kernel_separable<<<gridSize, blockSize>>>(d_blurred_temp, d_blurred, d_gfilter, 
                                                  d_filter_size, 
                                                  numRows, numCols, false);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

/**
* Call to the median filter
*/
void median_filter(unsigned char* d_frame,
                     unsigned char* d_blurred,
                     size_t numRows, size_t numCols)
{

  const dim3 blockSize(THREAD_SIZE, THREAD_SIZE, 1);
  const dim3 gridSize(numRows / THREAD_SIZE + 1, numCols / THREAD_SIZE + 1, 1); 
  // once in the x direction
  //default median filter size 3
  median_filter_kernel<<<gridSize, blockSize>>>(d_frame, d_blurred, numRows, numCols);

  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

/**
* A sequenced call to either the separable gaussian filter or the 2d filter and a subsequent call
* to the median filter CUDA kernels to run on the GPU with the device memory pointers provided
*/
void gaussian_and_median_blur(unsigned char* d_frame,
                     unsigned char* d_blurred,
                     unsigned char* d_blurred_temp,
                     const float* const d_gfilter,
                     size_t d_filter_size,
                     size_t numRows, size_t numCols)
{

  const dim3 blockSize(THREAD_SIZE, THREAD_SIZE, 1);
  const dim3 gridSize(numRows / THREAD_SIZE + 1, numCols / THREAD_SIZE + 1, 1); 

  #if SEPARATED_GAUSSIAN_FILTER == 1
  // once in the x direction
  gaussian_filter_kernel_separable<<<gridSize, blockSize>>>(d_frame, d_blurred, d_gfilter, 
                                                  d_filter_size, 
                                                  numRows, numCols, true);

  //once in the y direction
  gaussian_filter_kernel_separable<<<gridSize, blockSize>>>(d_blurred, d_blurred_temp, d_gfilter, 
                                                  d_filter_size, 
                                                  numRows, numCols, false);
  #else
  // in this case, also need to make sure the filter is 2d
  gaussian_filter_kernel<<<gridSize, blockSize>>>(d_frame, d_blurred_temp, d_gfilter, 
                                                  d_filter_size, d_filter_size, 
                                                  numRows, numCols);
  #endif

  median_filter_kernel<<<gridSize, blockSize>>>(d_blurred_temp, d_blurred, numRows, numCols);

  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
















//###################################################################################################################################################
//------------------------------------------------------------HOG-Feature Section--------------------------------------------------------------------------
//###################################################################################################################################################

//------------------------------------------------------------Includes--------------------------------------------------------------------------
#include <stdio.h>
#include <assert.h>

#include "hip/hip_runtime.h"
#include ""

#include <string.h>

// CUDA stuff:
#include "hip/hip_runtime.h"

// OpenCV stuff (note: C++ not C):
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

//------------------------------------------------------------HogSupport.cu--------------------------------------------------------------------------

hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    // We should be free()ing CPU+GPU memory here, but we're relying on the OS
    // to do it for us.
    hipDeviceReset();
    assert(result == hipSuccess);
  }
  return result;
}

void WriteNumbers(char* filename, float *features, int row, int col, int numbins)
{
	int i,j;
	//unsigned long int numbers[8192];
	FILE* f = fopen(filename, "w");
	if(f == NULL)
	{
		printf("\n\n%s NOT FOUND\n\n",filename);
		exit(1);
	}
	
	for (i=0;i<row*col;i++)
	{
		for(j=0;j<numbins;j++) {
      			if(numbins==4) fprintf(f, "%f ", features[i*numbins+j]);
		  	else fprintf(f, "%e ", features[i*numbins+j]);
		}
   	fprintf(f,"\n");
	}
}

//------------------------------------------------------------hotSupport.h--------------------------------------------------------------------------

//Cuda support programs

struct HogProp
{
	int ImgRow,ImgCol;
	int CellRow,CellCol;
  int TotalCells;
  int BlockRow,BlockCol;
  int TotalBlocks;
  int FeatureSize;
  int ImgSize;
	int CellSize;
	int BlockSize;
	int BlockOverlap;
	int NumBins;
	int Orientation;
};

struct DisplayProp
{
	int ImgRow,ImgCol;
	int CellRow,CellCol;
  int TotalCells;
  int HorzCells;
  int HorzCellsTotal;
  //int BlockRow,BlockCol;
  //int TotalBlocks;
  //int FeatureSize;
  int ImgSize;
	int CellSize;
	//int BlockSize;
	//int BlockOverlap;
	int NumBins;
  int DisplayCellSize;
  int DisplayImgRow,DisplayImgCol,DisplayImgSize;
	//int Orientation;
};

#define PI 3.141592654
//extern struct HogProp hp;

hipError_t checkCuda(hipError_t result);
int checkerror(int argc, char *argv[], struct HogProp hp);
void WriteNumbers(char* filename, float *features, int row, int col, int numbins);






//------------------------------------------------------------main.cu--------------------------------------------------------------------------





#define BOX_SIZE 8
hipError_t launch_helper(float* Runtimes);

struct HogProp hp;
struct DisplayProp dp;
uchar * CPU_InputArray, * CPU_OutputArray;
float *CPU_CellArray,*CPU_FeatureArray, *CPU_Hist;

hipStream_t stream[2];

using namespace cv;

int Cal_kernel_v;
int Cell_kernel_v;
int Block_kernel_v;
int Display_Cell_kernel_v;
int display_kernel_v;

bool firstRun = true;

//-------------------------------------------------------------Cal_kernel-------------------------------------------------------------------------

// Cal_kernel Original Version 0
__global__ void Cal_kernel_v0(uchar *GPU_i, int *Orientation,float *Gradient, uchar *DisplayOrientation, HogProp hp){
 	int i = blockIdx.x * blockDim.x + threadIdx.x;  // row of image
	int j = blockIdx.y * blockDim.y + threadIdx.y;  // col of image
   
  float ang,displayang;
	float gx,gy;
  int idx = i*hp.ImgCol + j;
  int idx_prev= (i-1)*hp.ImgCol + j;
  int idx_next= (i+1)*hp.ImgCol + j;
 
  if(i>0 && i < hp.ImgRow-1 && j >0 && j < hp.ImgCol-1){
   	gx=(float)(GPU_i[idx-1]-GPU_i[idx+1]);
    gy=(float)(GPU_i[idx_prev]-GPU_i[idx_next]);
    
     Gradient[idx]=sqrtf(gx*gx+gy*gy);
     ang= atan2f(gy,gx);
     
     if(ang<0) {
       displayang=8*(ang+PI);
     }
     else displayang=8*ang;
     
     if(displayang<PI | displayang>7*PI)          DisplayOrientation[idx]=0;
     else if(displayang>=PI & displayang<3*PI)    DisplayOrientation[idx]=1;
     else if(displayang>=3*PI & displayang<5*PI)  DisplayOrientation[idx]=2;
     else                                         DisplayOrientation[idx]=3;
          
     if (ang<0){
       if(hp.Orientation==0) { ang = ang+ PI; }
       else { ang = 2*PI+ang; }
     }
     
     if(hp.Orientation==0) ang=(hp.NumBins)*ang/PI;
     else ang=(hp.NumBins)*ang/(2*PI);
     
     Orientation[idx]=(int)ang;
     //GPU_o[idx] = (uchar) (DisplayOrientation[idx]);
  }
}

// Cal_kernel Optimized Version 1
// No significant execution time performance gains
// Replaced the C-style casting with static_cast to improve code safety and readability
// Merged the two separate branches for updating the displayang variable into a single line using the ternary operator, simplifying the control flow
__global__ void Cal_kernel_v1(uchar *GPU_i, int *Orientation,float *Gradient, uchar *DisplayOrientation, HogProp hp){
  // Calculate row and column indices for the current thread
  int i = blockIdx.x * blockDim.x + threadIdx.x; // row of image
  int j = blockIdx.y * blockDim.y + threadIdx.y; // col of image

  // Declare variables for gradient angle, display angle, and gradient components
  float ang, displayang;
  float gx, gy;
  // Calculate linear index for current, previous, and next rows in the image
  int idx = i * hp.ImgCol + j;
  int idx_prev = (i - 1) * hp.ImgCol + j;
  int idx_next = (i + 1) * hp.ImgCol + j;

  // Ensure the current thread is within image boundaries
  if (i > 0 && i < hp.ImgRow - 1 && j > 0 && j < hp.ImgCol - 1){
    // Compute gradient components using static_cast for type safety and readability
    gx = static_cast<float>(GPU_i[idx - 1] - GPU_i[idx + 1]);
    gy = static_cast<float>(GPU_i[idx_prev] - GPU_i[idx_next]);

    // Calculate gradient magnitude
    Gradient[idx] = sqrtf(gx * gx + gy * gy);
    // Calculate gradient angle
    ang = atan2f(gy, gx);

    // Calculate display angle using the ternary operator for simplified control flow
    displayang = (ang < 0) ? 8 * (ang + PI) : 8 * ang;

    // Assign display orientation based on the display angle
    if (displayang < PI || displayang > 7 * PI)
      DisplayOrientation[idx] = 0;
    else if (displayang >= PI && displayang < 3 * PI)
      DisplayOrientation[idx] = 1;
    else if (displayang >= 3 * PI && displayang < 5 * PI)
      DisplayOrientation[idx] = 2;
    else
      DisplayOrientation[idx] = 3;

    // Adjust the angle if it's negative
    if (ang < 0){
      if (hp.Orientation == 0){
        ang = ang + PI;
      }
      else{
        ang = 2 * PI + ang;
      }
    }

    // Calculate bin index for the current angle based on the selected orientation mode
    if (hp.Orientation == 0)
      ang = (hp.NumBins)*ang / PI;
    else
      ang = (hp.NumBins)*ang / (2 * PI);
    
    // Store the bin index in the Orientation array
    Orientation[idx] = static_cast<int>(ang);
  }

}

//-------------------------------------------------------------Cell_kernel-------------------------------------------------------------------------

// Cell_kernel Original Version 0
__global__ void Cell_kernel_v0(float *histogram, int *Orientation,float *Gradient, HogProp hp){
 	int i = blockIdx.x * blockDim.x + threadIdx.x;  // row of image
	int j = blockIdx.y * blockDim.y + threadIdx.y;  // col of image
   
  int idx = i*hp.ImgCol*hp.CellSize + j*hp.CellSize;
  int idcell = i*hp.CellCol*hp.NumBins + j*hp.NumBins;
  int current_i,m,n;
  //int idx_next= (i+1)*hp.ImgCol + j;
  
  if(i<hp.CellRow & j<hp.CellCol) {
    for (m=0;m<hp.CellSize;m++) {
      current_i=idx+m*hp.ImgCol;
      for (n=0;n<hp.CellSize;n++) {
        histogram[idcell+Orientation[current_i+n]]+=Gradient[current_i+n];
      }
    }
  }
}

// Cell_kernel Optimized Version 1
// Significantly improved the execution time performance compared to original version 0
// Modified the thread configuration by adding the z-dimension and setting threadsPerBlock.z equal to hp.CellSize * hp.CellSize
// Takes full advantage of the GPU's three-dimensional parallelism, distributing the workload more evenly across the threads and increasing the overall throughput
// In the original Cell_kernel, there were two nested loops, which contributed to a higher degree of sequential processing
// By leveraging the z-dimension of the threads and assigning each thread to a unique combination of (m, n) indices, we were able to eliminate the need for these nested loops
// The optimized kernel has a more coherent memory access pattern, as each thread accesses consecutive memory locations when reading from and writing to the Orientation and Gradient arrays
// This improvement in memory access pattern reduces memory latency and contributes to the overall performance improvement
__global__ void Cell_kernel_v1(float *histogram, int *Orientation,float *Gradient, HogProp hp){
  // Calculate row, column, and cell indices for the current thread
  int i = blockIdx.x * blockDim.x + threadIdx.x; // row of image
  int j = blockIdx.y * blockDim.y + threadIdx.y; // col of image
  int k = threadIdx.z; // index within the cell

  // Ensure the current thread is within image and cell boundaries
  if (i < hp.CellRow && j < hp.CellCol && k < hp.CellSize * hp.CellSize){
    // Calculate cell_i and cell_j, which represent the local row and column indices within the cell
    int cell_i = k / hp.CellSize;
    int cell_j = k % hp.CellSize;

    // Calculate the global row and column indices in the image corresponding to the current thread
    int img_i = i * hp.CellSize + cell_i;
    int img_j = j * hp.CellSize + cell_j;

    // Calculate the linear indices for the image and the cell histogram
    int img_idx = img_i * hp.ImgCol + img_j;
    int cell_idx = i * hp.CellCol * hp.NumBins + j * hp.NumBins;

    // Update the cell histogram by accumulating gradient values based on their orientation
    histogram[cell_idx + Orientation[img_idx]] += Gradient[img_idx];
  }
}

// Cell_kernel Optimized Version 2
// Significantly improved the execution time performance compared to original version 0, and slightly improved from version 1
// Adds shared memory utilization to store the portion of Orientation and Gradient arrays required by each thread
// Reduces the number of global memory accesses
// Also includes the optimization methods of Version 1 described below
// Modified the thread configuration by adding the z-dimension and setting threadsPerBlock.z equal to hp.CellSize * hp.CellSize
// Takes full advantage of the GPU's three-dimensional parallelism, distributing the workload more evenly across the threads and increasing the overall throughput
// In the original Cell_kernel, there were two nested loops, which contributed to a higher degree of sequential processing
// By leveraging the z-dimension of the threads and assigning each thread to a unique combination of (m, n) indices, we were able to eliminate the need for these nested loops
// The optimized kernel has a more coherent memory access pattern, as each thread accesses consecutive memory locations when reading from and writing to the Orientation and Gradient arrays
// This improvement in memory access pattern reduces memory latency and contributes to the overall performance improvement
__global__ void Cell_kernel_v2(float *histogram, int *Orientation,float *Gradient, HogProp hp){
  // Calculate row, column, and cell indices for the current thread
  int i = blockIdx.x * blockDim.x + threadIdx.x; // row of image
  int j = blockIdx.y * blockDim.y + threadIdx.y; // col of image
  int k = threadIdx.z; // index within the cell

  // Allocate shared memory for Orientation and Gradient data
  __shared__ int shared_Orientation[BOX_SIZE][BOX_SIZE];
  __shared__ float shared_Gradient[BOX_SIZE][BOX_SIZE];

  // Ensure the current thread is within image and cell boundaries
  if (i < hp.CellRow && j < hp.CellCol && k < hp.CellSize * hp.CellSize) {
    // Calculate cell_i and cell_j, which represent the local row and column indices within the cell
    int cell_i = k / hp.CellSize;
    int cell_j = k % hp.CellSize;

    // Calculate the global row and column indices in the image corresponding to the current thread
    int img_i = i * hp.CellSize + cell_i;
    int img_j = j * hp.CellSize + cell_j;

    // Calculate the linear indices for the image and the cell histogram
    int img_idx = img_i * hp.ImgCol + img_j;
    int cell_idx = i * hp.CellCol * hp.NumBins + j * hp.NumBins;

    // Load data from global memory to shared memory
    shared_Orientation[cell_i][cell_j] = Orientation[img_idx];
    shared_Gradient[cell_i][cell_j] = Gradient[img_idx];

    // Ensure that all data is loaded into shared memory before proceeding
    __syncthreads();

    // Perform the histogram computation using shared memory arrays
    histogram[cell_idx + shared_Orientation[cell_i][cell_j]] += shared_Gradient[cell_i][cell_j];
  }
}

//-------------------------------------------------------------Block_kernel-------------------------------------------------------------------------

// Block_kernel Original Version 0
__global__ void Block_kernel_v0(float *FinalFeatures, float *histogram, HogProp hp){

 	int i = blockIdx.x * blockDim.x + threadIdx.x;  // row of image
	int j = blockIdx.y * blockDim.y + threadIdx.y;  // col of image
  
  int step=hp.BlockSize-hp.BlockOverlap;
  int idblock = i*hp.BlockCol*hp.FeatureSize + j*hp.FeatureSize;
  int idcell = i*hp.CellCol*step*hp.NumBins + j*step*hp.NumBins;
  int current_i,current_j,m,n;
  float average=0.000000001;
  int horz=hp.BlockSize*hp.NumBins;
  //int idx_next= (i+1)*hp.ImgCol + j;
  
  if(i<hp.BlockRow & j<hp.BlockCol) {
    for (m=0;m<hp.BlockSize;m++) {
      current_i=idcell+m*hp.CellCol*hp.NumBins;
      for (n=0;n<horz;n++) {
        average=average+histogram[current_i+n];
      }
    }
  }
  
  if(i<hp.BlockRow & j<hp.BlockCol) {
    for (m=0;m<hp.BlockSize;m++) {
      current_i=idcell+m*hp.CellCol*hp.NumBins;
      current_j=idblock+m*hp.CellCol;
      for (n=0;n<horz;n++) {
        FinalFeatures[current_j+n]=histogram[current_i+n]/average;
      }
    }
  }
}

// Block_kernel Optimized Version 1
// Seemed to increase the execution time slightly giving worse performance than Version 0
// Reduction of redundant memory accesses by utilizing shared memory to store histogram values
// Merged the two separate loops into a single loop to streamline the code
__global__ void Block_kernel_v1(float *FinalFeatures, float *histogram, HogProp hp){
  // Calculate row and column indices for the current thread
  int i = blockIdx.x * blockDim.x + threadIdx.x; // row of image
  int j = blockIdx.y * blockDim.y + threadIdx.y; // col of image

  // Calculate the step size, idblock, and idcell for the current thread
  int step = hp.BlockSize - hp.BlockOverlap;
  int idblock = i * hp.BlockCol * hp.FeatureSize + j * hp.FeatureSize;
  int idcell = i * hp.CellCol * step * hp.NumBins + j * step * hp.NumBins;

  // Initialize loop variables and the average value
  int current_i, current_j, m, n;
  float average = 0.000000001f;
  int horz = hp.BlockSize * hp.NumBins;

  // Allocate shared memory for histogram values
  __shared__ float shHistogram[256]; // assuming the maximum value of hp.BlockSize * hp.NumBins is 256
  int threadId = threadIdx.x * blockDim.y + threadIdx.y;

  // Ensure the current thread is within the block row and column boundaries
  if (i < hp.BlockRow && j < hp.BlockCol){
    // Combine the previously separate loops into a single loop for streamlined code
    for (m = 0; m < hp.BlockSize; m++){
      // Calculate the current_i and current_j values for the loop iteration
      current_i = idcell + m * hp.CellCol * hp.NumBins;
      current_j = idblock + m * hp.CellCol;

      // Load histogram values into shared memory and compute the final features
      for (n = 0; n < horz; n++){
        // Load histogram values into shared memory if threadId is within the valid range
        if (threadId < 256){
          shHistogram[threadId] = histogram[current_i + n];
        }
        __syncthreads(); // Ensure all threads have loaded the data before proceeding

        // Update the average value using the shared memory histogram values
        average += shHistogram[threadId];
        __syncthreads(); // Ensure all threads have updated the average value before proceeding

        // Compute the final feature value and store it in FinalFeatures
        FinalFeatures[current_j + n] = shHistogram[threadId] / average;
      }
    }
  }
}

// Block_kernel Optimized Version 2
// Improved execution time compared to original version 0 and 1
// Using global memory to optimize the kernel by reducing the number of calculations inside the loop
// Calculates the inverse of the average value outside the loop and then multiplies it with the histogram values to normalize them
__global__ void Block_kernel_v2(float *FinalFeatures, float *histogram, HogProp hp){
  // Calculate row and column indices for the current thread
  int i = blockIdx.x * blockDim.x + threadIdx.x;  // row of image
  int j = blockIdx.y * blockDim.y + threadIdx.y;  // col of image

  // Calculate the step size, idblock, and idcell for the current thread
  int step = hp.BlockSize - hp.BlockOverlap;
  int idblock = i * hp.BlockCol * hp.FeatureSize + j * hp.FeatureSize;
  int idcell = i * hp.CellCol * step * hp.NumBins + j * step * hp.NumBins;

  // Initialize loop variables and the average value
  int current_i, current_j, m, n;
  float average = 0.000000001;
  int horz = hp.BlockSize * hp.NumBins;

  // Ensure the current thread is within the block row and column boundaries
  if (i < hp.BlockRow && j < hp.BlockCol) {
    // Compute the average value using the histogram values from global memory
    for (m = 0; m < hp.BlockSize; m++) {
      current_i = idcell + m * hp.CellCol * hp.NumBins;
      for (n = 0; n < horz; n++) {
        average = average + histogram[current_i + n];
      }
    }

    // Calculate the inverse of the average value
    float inv_average = 1.0f / average;

    // Normalize the histogram values using the inverse of the average value and store the result in FinalFeatures
    for (m = 0; m < hp.BlockSize; m++) {
      current_i = idcell + m * hp.CellCol * hp.NumBins;
      current_j = idblock + m * hp.CellCol;
      for (n = 0; n < horz; n++) {
        FinalFeatures[current_j + n] = histogram[current_i + n] * inv_average;
      }
    }
  }
}

// Block_kernel Optimized Version 3
// Applies the Subhistogram accumulation method to the Block_kernel
// Reduced global memory accesses
// Each thread is assigned to a unique index in the final histogram, there are no conflicts or need for atomic operations
// No synchronization needed, there are no shared memory dependencies or conflicts that would necessitate synchronization among threads
// The optimized Version 3 Block_kernel has a simpler code structure
// Improved performance significantly vs. Version 0 and also improved vs. Version 1 and 2
__global__ void Block_kernel_v3(float *FinalFeatures, float *histogram, HogProp hp){
  // Calculate the block and thread indices for the current thread
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Calculate the unique blockId, cellIdX, and cellIdY for the current thread
  int blockId = by * hp.BlockCol + bx;
  int cellIdX = bx * (hp.BlockSize - hp.BlockOverlap) + tx;
  int cellIdY = by * (hp.BlockSize - hp.BlockOverlap) + ty;

  // Ensure the current thread is within the cell row and column boundaries
  if (cellIdX < hp.CellCol && cellIdY < hp.CellRow){
    // Calculate the unique cellId and histIndex for the current thread
    int cellId = cellIdY * hp.CellCol + cellIdX;
    int histIndex = blockId * hp.FeatureSize + ty * hp.CellCol * hp.NumBins + tx * hp.NumBins;

    // Accumulate the subhistograms into the final histogram
    for (int i = 0; i < hp.NumBins; i++){
      FinalFeatures[histIndex + i] += histogram[cellId * hp.NumBins + i];
    }
  }
}

//-------------------------------------------------------------Display_Cell_kernel-------------------------------------------------------------------------

// Display_Cell_kernel Original Version 0
__global__ void Display_Cell_kernel_v0(float* Displayhistogram, float *TempDisplayhistogram, uchar *DisplayOrientation,float *Gradient, DisplayProp dp){
  int i = blockIdx.x * blockDim.x + threadIdx.x;  // row of image
	int j = blockIdx.y * blockDim.y + threadIdx.y;  // col of image
   
  int idx = i*dp.ImgCol + j*dp.CellSize;
  int idxtemp = i*dp.CellCol*dp.NumBins*dp.CellSize + j*dp.NumBins;
  int idcell = i*dp.CellCol*dp.NumBins + j*dp.NumBins;
  int n;
  int temp_rowsize=dp.CellCol*dp.NumBins;
  //float avg;
  float max1,max2,avg;
  //int idx_next= (i+1)*hp.ImgCol + j;
  
  if(i<dp.HorzCells & j<dp.CellCol) {
    TempDisplayhistogram[idcell]=0; TempDisplayhistogram[idcell+1]=0; TempDisplayhistogram[idcell+2]=0; TempDisplayhistogram[idcell+3]=0;
    for (n=0;n<dp.CellSize;n++) {
      TempDisplayhistogram[idcell+DisplayOrientation[idx+n]]+=Gradient[idx+n];
    }
  }
  
  __syncthreads();
  
  if(i<dp.CellRow) {
    for(n=0;n<dp.CellSize;n++) {
      Displayhistogram[idcell]+=TempDisplayhistogram[idxtemp+n*temp_rowsize];
      Displayhistogram[idcell+1]+=TempDisplayhistogram[idxtemp+n*temp_rowsize+1];
      Displayhistogram[idcell+2]+=TempDisplayhistogram[idxtemp+n*temp_rowsize+2];
      Displayhistogram[idcell+3]+=TempDisplayhistogram[idxtemp+n*temp_rowsize+3];
    }
    
    if(Displayhistogram[idcell]>Displayhistogram[idcell+1]) {max1=Displayhistogram[idcell];}   else {max1=Displayhistogram[idcell+1];}
    if(Displayhistogram[idcell+2]>Displayhistogram[idcell+3]) {max2=Displayhistogram[idcell+2];} else {max2=Displayhistogram[idcell+3];}
    if(max2>max1) max1=max2;
    avg=max1/8;
    //avg=(Displayhistogram[idcell+3]+Displayhistogram[idcell+2]+Displayhistogram[idcell+1]+Displayhistogram[idcell])/8;
    //avg=1;
    if(Displayhistogram[idcell+3]>=0) Displayhistogram[idcell+3]=Displayhistogram[idcell+3]/avg; else Displayhistogram[idcell+3]=0;
    if(Displayhistogram[idcell+2]>=0) Displayhistogram[idcell+2]=Displayhistogram[idcell+2]/avg; else Displayhistogram[idcell+2]=0;
    if(Displayhistogram[idcell+1]>=0) Displayhistogram[idcell+1]=Displayhistogram[idcell+1]/avg; else Displayhistogram[idcell+1]=0;
    if(Displayhistogram[idcell]>=0) Displayhistogram[idcell]=Displayhistogram[idcell]/avg; else Displayhistogram[idcell]=0;
  }
}

// Display_Cell_kernel Optimized Version 1
// No changes in performance execution time
// Reducing redundant calculations and removing unnecessary branches
// Combined the initialization of TempDisplayhistogram elements with the first loop, removing the need for a separate loop
// Replaced the branches with the use of fmaxf function to find the maximum values and updated the normalization step to use a single loop
__global__ void Display_Cell_kernel_v1(float* Displayhistogram, float *TempDisplayhistogram, uchar *DisplayOrientation,float *Gradient, DisplayProp dp){
  int i = blockIdx.x * blockDim.x + threadIdx.x;  // row of image
  int j = blockIdx.y * blockDim.y + threadIdx.y;  // col of image

  int idx = i * dp.ImgCol + j * dp.CellSize;
  int idxtemp = i * dp.CellCol * dp.NumBins * dp.CellSize + j * dp.NumBins;
  int idcell = i * dp.CellCol * dp.NumBins + j * dp.NumBins;
  int n;
  int temp_rowsize = dp.CellCol * dp.NumBins;
  float max1, max2, avg;

  // Initialize TempDisplayhistogram elements and calculate histogram
  if (i < dp.HorzCells && j < dp.CellCol) {
    for (int k = 0; k < dp.NumBins; k++) {
      TempDisplayhistogram[idcell + k] = 0;
    }

    for (n = 0; n < dp.CellSize; n++) {
      TempDisplayhistogram[idcell + DisplayOrientation[idx + n]] += Gradient[idx + n];
    }
  }

  __syncthreads();

  // Combine histograms and normalize
  if (i < dp.CellRow) {
    for (n = 0; n < dp.CellSize; n++) {
      for (int k = 0; k < dp.NumBins; k++) {
        Displayhistogram[idcell + k] += TempDisplayhistogram[idxtemp + n * temp_rowsize + k];
      }
    }

    // Calculate the maximum value using fmaxf function
    max1 = fmaxf(Displayhistogram[idcell], Displayhistogram[idcell + 1]);
    max2 = fmaxf(Displayhistogram[idcell + 2], Displayhistogram[idcell + 3]);
    avg = fmaxf(max1, max2) / 8;

    // Normalize the histogram values
    for (int k = 0; k < dp.NumBins; k++) {
      Displayhistogram[idcell + k] = fmaxf(Displayhistogram[idcell + k] / avg, 0);
    }
  }
}

//-------------------------------------------------------------display_kernel-------------------------------------------------------------------------

// display_kernel Original Version 0
__global__ void display_kernel_v0(float *Displayhistogram, uchar *GPU_odata, DisplayProp dp){
 	int i = blockIdx.x * blockDim.x + threadIdx.x;  // row of image
	int j = blockIdx.y * blockDim.y + threadIdx.y;  // col of image
  int k = threadIdx.z;
   
  int idx = i*dp.CellCol*4 + j*4+k;
  int idcell = i*dp.DisplayCellSize*dp.DisplayImgCol + j*dp.DisplayCellSize;
  int m;
  int temp=(int)Displayhistogram[idx];
  int tempid;
  
  tempid=idcell+8+8*dp.DisplayImgCol;
  for(m=1;m<temp ;m++) {
    if(k==0) {
      GPU_odata[tempid+m]=255; GPU_odata[tempid-m]=255;
    }else if(k==1) {
      GPU_odata[tempid+m-m*dp.DisplayImgCol]=255; GPU_odata[tempid-m+m*dp.DisplayImgCol]=255;
    }else if(k==2) {
      GPU_odata[tempid-m*dp.DisplayImgCol]=255; GPU_odata[tempid+m*dp.DisplayImgCol]=255;
    }else {
      GPU_odata[tempid+m+m*dp.DisplayImgCol]=255; GPU_odata[tempid+m+m*dp.DisplayImgCol]=255;
    }
  }
  if(k==0) GPU_odata[tempid]=255;
}

// display_kernel Optimized Version 1
// Instead of having a separate if block for each value of k, 
// A single block is used that computes the offset based on the value of k
// This will reduce the number of branches in the code, which may help improve performance
// Did not provide any performance changes
__global__ void display_kernel_v1(float *Displayhistogram, uchar *GPU_odata, DisplayProp dp){
  int i = blockIdx.x * blockDim.x + threadIdx.x;  // row of image
  int j = blockIdx.y * blockDim.y + threadIdx.y;  // col of image
  int k = threadIdx.z;
  
  int idx = i * dp.CellCol * 4 + j * 4 + k;
  int idcell = i * dp.DisplayCellSize * dp.DisplayImgCol + j * dp.DisplayCellSize;
  int m;
  int temp = (int)Displayhistogram[idx];
  int tempid;

  tempid = idcell + 8 + 8 * dp.DisplayImgCol;

  // Define offset arrays for x and y based on the value of k
  int offset_x[] = {1, 1, 0, -1};
  int offset_y[] = {0, 1, 1, 1};

  // Draw lines for each value of k using the offset arrays
  for(m = 1; m < temp; m++) {
    int x1 = tempid + m * offset_x[k];
    int x2 = tempid - m * offset_x[k];
    int y1 = m * offset_y[k];
    int y2 = -m * offset_y[k];

    GPU_odata[x1 + y1 * dp.DisplayImgCol] = 255;
    GPU_odata[x2 + y2 * dp.DisplayImgCol] = 255;
  }
  
  // Set the center pixel to 255 for k == 0
  if(k == 0) GPU_odata[tempid] = 255;
}

// display_kernel Optimized Version 2
// Moved the calculations of the offsets outside of the loop and used accumulative sums to reduce the number of calculations inside the loop
// Did not provide any performance changes
__global__ void display_kernel_v2(float *Displayhistogram, uchar *GPU_odata, DisplayProp dp){
  // Calculate the row, column, and bin index for the current thread
  int i = blockIdx.x * blockDim.x + threadIdx.x;  // row of image
  int j = blockIdx.y * blockDim.y + threadIdx.y;  // col of image
  int k = threadIdx.z;  // bin index

  // Calculate the histogram index and display cell index for the current thread
  int idx = i * dp.CellCol * 4 + j * 4 + k;
  int idcell = i * dp.DisplayCellSize * dp.DisplayImgCol + j * dp.DisplayCellSize;

  int m;  // loop variable for iterating through the histogram values
  int temp = (int)Displayhistogram[idx];  // Get the histogram value at the current index
  int tempid;  // temporary variable to store the pixel index in the display image

  // Calculate the starting index in the display image
  tempid = idcell + 8 + 8 * dp.DisplayImgCol;

  // Define the x and y offset arrays based on the bin index (k)
  int offset_x[] = {1, 1, 0, -1};
  int offset_y[] = {0, 1, 1, 1};

  // Initialize base values for x and y coordinates
  int base_x1 = tempid;
  int base_x2 = tempid;
  int base_y1 = 0;
  int base_y2 = 0;

  // Iterate through the values of m and use accumulative sums for x and y coordinates
  for(m = 1; m < temp; m++) {
    // Update the x and y coordinates using the accumulative sums
    base_x1 += offset_x[k];
    base_x2 -= offset_x[k];
    base_y1 += offset_y[k];
    base_y2 -= offset_y[k];

    // Set the corresponding pixels in the display image to 255 (white) based on the updated x and y coordinates
    GPU_odata[base_x1 + base_y1 * dp.DisplayImgCol] = 255;
    GPU_odata[base_x2 + base_y2 * dp.DisplayImgCol] = 255;
  }
  
  // Set the center pixel to 255 (white) for k == 0
  if(k == 0) GPU_odata[tempid] = 255;
}

void writeImageToFile(Mat image){
  if (!imwrite("HOG-Feature_output.bmp", image)) {
		fprintf(stderr, "couldn't write output to disk!\n");
		hipHostFree(CPU_OutputArray);
    hipHostFree(CPU_InputArray);
	  hipHostFree(CPU_FeatureArray);
		exit(EXIT_FAILURE);
	}
}

// hogFeature takes in Mat image and returns the Mat image of the HOG features extracted
Mat hogFeature(Mat image){
	
	//-------------------------------------------------------------variables-------------------------------------------------------------------------
	//int i;
  float GPURuntimes[4];
	//===============================================================================================================================================

  // Setting input parameters adjusted optimized performance set
  hp.CellSize= 8; //atoi(argv[3]); 
  hp.BlockSize= 8; //atoi(argv[4]); 
  hp.BlockOverlap= 0; //atoi(argv[5]); 		  
  hp.NumBins= 9; //atoi(argv[6]); 
  hp.Orientation= 0; //atoi(argv[7]);

  // Using optimized kernels versions for performance set
  Cal_kernel_v = 0; //atoi(argv[3]);
  Cell_kernel_v = 0; //atoi(argv[4]);
  Block_kernel_v = 0; //atoi(argv[5]);
  Display_Cell_kernel_v = 0; //atoi(argv[6]);
  display_kernel_v = 0; //atoi(argv[7]);

	if(! image.data ) {
		fprintf(stderr, "Could not open or find the image.\n");
		exit(EXIT_FAILURE);
	}

	hp.ImgRow=image.rows;
	hp.ImgCol=image.cols;
  hp.ImgSize=hp.ImgRow*hp.ImgCol;
  hp.CellRow=floor(image.rows/hp.CellSize);
  hp.CellCol=floor(image.cols/hp.CellSize);
  hp.TotalCells=hp.CellRow*hp.CellCol;
	hp.BlockRow=(hp.CellRow-hp.BlockSize+1)/(hp.BlockSize-hp.BlockOverlap);
  hp.BlockCol=(hp.CellCol-hp.BlockSize+1)/(hp.BlockSize-hp.BlockOverlap);
  hp.TotalBlocks=hp.BlockRow*hp.BlockCol;
  hp.FeatureSize=hp.NumBins*hp.BlockSize*hp.BlockSize;

  // Comment printf out for project
  // printf("----------------------------------IMAGE DIVIDED INTO CELL HISTOGRAM----------------\n");
  // printf("\nCell_rows = %d, Cell_columns = %d, Total_cells = %d\n",hp.CellRow,hp.CellCol,hp.TotalCells);
	// printf("\nBlock_rows = %d, Block_columns = %d, Total_blocks = %d\n",hp.BlockRow,hp.BlockCol,hp.TotalBlocks);
  // printf("\nfeaturesize=%d\n",hp.FeatureSize);
  // printf("-----------------------------------------------------------------------------------\n\n");
  
  dp.ImgRow=hp.ImgRow;
  dp.ImgCol=hp.ImgCol;
  dp.ImgSize=hp.ImgSize;
  dp.CellRow=32;
  dp.CellSize=dp.ImgRow/dp.CellRow;
  dp.CellCol=dp.ImgCol/dp.CellSize;
  dp.TotalCells=dp.CellRow*dp.CellCol;
  dp.NumBins=4;
  dp.HorzCellsTotal=dp.CellSize*dp.TotalCells;
  dp.HorzCells=dp.CellSize*dp.CellRow;
  
  dp.DisplayCellSize=17;
  dp.DisplayImgRow=dp.DisplayCellSize*dp.CellRow;
  dp.DisplayImgCol=dp.DisplayCellSize*dp.CellCol;
  dp.DisplayImgSize=dp.DisplayImgCol*dp.DisplayImgRow;

  // Comment printf out for project
  // printf("----------------------IMAGE DIVIDED INTO CELL HISTOGRAM FOR DISPLAYING-------------\n");
  // printf("\nCell_rows = %d, Cell_columns = %d, Total_cells=%d, Cell_size=%d, Horz_cells=%d\n",dp.CellRow,dp.CellCol,dp.TotalCells,dp.CellSize,dp.HorzCells);
  // printf("\nDisplay_rows = %d, Display_columns = %d, Total_pixels=%d, Cell_size=%d\n",dp.DisplayImgRow,dp.DisplayImgCol,dp.DisplayImgSize,dp.DisplayCellSize);
  // printf("-----------------------------------------------------------------------------------\n\n");

  //===============================================================================================================================================	

	//---------------------------------------------------Create CPU memory to store the output-------------------------------------------------------
	
  checkCuda(hipHostMalloc ((void**)&CPU_InputArray,hp.ImgSize));
  checkCuda(hipHostMalloc ((void**)&CPU_OutputArray,dp.DisplayImgSize));	
  checkCuda(hipHostMalloc ((void**)&CPU_Hist,dp.TotalCells *4*4));	
  checkCuda(hipHostMalloc ((void**)&CPU_FeatureArray,hp.TotalBlocks*sizeof(float)*hp.FeatureSize));	
  memcpy(CPU_InputArray,image.data,hp.ImgSize);
  checkCuda(launch_helper(GPURuntimes));

  // Output the HOG features to the SVM classifier 
  Mat hogFeatureOutput = Mat(dp.DisplayImgRow, dp.DisplayImgCol, CV_8UC1, CPU_InputArray);

  if(firstRun){
    printf("----------------------------------HOG Feature Timings----------------\n");
	  printf("Tfr CPU->GPU = %5.2f ms ... \nExecution = %5.2f ms ... \nTfr GPU->CPU = %5.2f ms   \n Total=%5.2f ms\n", GPURuntimes[1], GPURuntimes[2], GPURuntimes[3], GPURuntimes[0]);
    printf("-----------------------------------------------------------------------------------\n\n");
    writeImageToFile(hogFeatureOutput);
    firstRun = false;
  }

  hipHostFree(CPU_OutputArray);
  hipHostFree(CPU_InputArray);
  hipHostFree(CPU_Hist);	
	hipHostFree(CPU_FeatureArray);

  return hogFeatureOutput; // return the HOG Feature image output for the SVM
}

hipError_t launch_helper(float* Runtimes){
	hipEvent_t time1, time2, time3, time4;

  int   *Orientation;
	float *Gradient;
  uchar *DisplayOrientation;
	uchar *GPU_idata;
	uchar *GPU_odata;
 	//uchar *GPU_displaydata;
  float *GPU_CellHistogram;
  float *GPU_BlockHistogram;
  float *TempDisplayhistogram;
  float *Displayhistogram;
  dim3 threadsPerBlock;
	dim3 numBlocks;
  int i;
  
  hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);  
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}
 
  hipEventCreate(&time1);
  hipEventCreate(&time2);
  hipEventCreate(&time3);
  hipEventCreate(&time4);
 
 hipEventRecord(time1, 0);
 
 for(i=0;i<2;i++) checkCuda(hipStreamCreate(&stream[i]));
 
 checkCuda(hipMalloc((void**)&GPU_idata, hp.ImgSize));
 checkCuda(hipMalloc((void**)&Gradient, hp.ImgSize*4));
 checkCuda(hipMalloc((void**)&Orientation, hp.ImgSize*4));
 checkCuda(hipMalloc((void**)&DisplayOrientation, hp.ImgSize));
 checkCuda(hipMemcpyAsync(GPU_idata, CPU_InputArray, hp.ImgSize, hipMemcpyHostToDevice,stream[0]));
 hipEventRecord(time2, 0);

 //-------------------------------------------------------------Cal_kernel-------------------------------------------------------------------------
 threadsPerBlock = dim3(BOX_SIZE, BOX_SIZE);
 numBlocks = dim3((int)ceil(hp.ImgRow / (float)threadsPerBlock.x), (int)ceil(hp.ImgCol / (float)threadsPerBlock.y));

 if(Cal_kernel_v==1){
  Cal_kernel_v1<<<numBlocks, threadsPerBlock,0,stream[0]>>>(GPU_idata,Orientation,Gradient,DisplayOrientation,hp);
 } else {
  Cal_kernel_v0<<<numBlocks, threadsPerBlock,0,stream[0]>>>(GPU_idata,Orientation,Gradient,DisplayOrientation,hp);
 }

 checkCuda(hipDeviceSynchronize());
 // $$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$ hipFree(GPU_idata);
 
 //-------------------------------------------------------------Display_Cell_kernel-------------------------------------------------------------------------
 checkCuda(hipMalloc((void**)&TempDisplayhistogram, dp.HorzCellsTotal*4*4));
 checkCuda(hipMalloc((void**)&Displayhistogram, dp.TotalCells *4*4)); 

 // The original Display_Cell_kernel
 threadsPerBlock = dim3(BOX_SIZE, BOX_SIZE);
 numBlocks = dim3((int)ceil(dp.HorzCells / (float)threadsPerBlock.x), (int)ceil(dp.CellCol / (float)threadsPerBlock.y));

 if(Display_Cell_kernel_v==1){
  Display_Cell_kernel_v1<<<numBlocks, threadsPerBlock,0,stream[1]>>>(Displayhistogram,TempDisplayhistogram,DisplayOrientation,Gradient,dp);
 } else {
  Display_Cell_kernel_v0<<<numBlocks, threadsPerBlock,0,stream[1]>>>(Displayhistogram,TempDisplayhistogram,DisplayOrientation,Gradient,dp);
 }
 
 //-------------------------------------------------------------Cell_kernel-------------------------------------------------------------------------
 // Cell_kernel
 checkCuda(hipHostMalloc ((void**)&GPU_CellHistogram,hp.TotalCells*sizeof(float)*hp.NumBins));

 // Call the kernel
 if(Cell_kernel_v==1){
  // Optimized Cell_kernel 3D
  // Update threadsPerBlock to include the hp.CellSize in the z-dimension
  threadsPerBlock.x = BOX_SIZE;
  threadsPerBlock.y = BOX_SIZE;
  threadsPerBlock.z = hp.CellSize * hp.CellSize;

  // Update numBlocks definition
  numBlocks.x = (int)ceil(hp.CellRow / (float)threadsPerBlock.x);
  numBlocks.y = (int)ceil(hp.CellCol / (float)threadsPerBlock.y);

  Cell_kernel_v1<<<numBlocks, threadsPerBlock, 0, stream[0]>>>(GPU_CellHistogram, Orientation, Gradient, hp);
 } else if(Cell_kernel_v==2){
  // Optimized Cell_kernel 3D
  // Update threadsPerBlock to include the hp.CellSize in the z-dimension
  threadsPerBlock.x = BOX_SIZE;
  threadsPerBlock.y = BOX_SIZE;
  threadsPerBlock.z = hp.CellSize * hp.CellSize;

  // Update numBlocks definition
  numBlocks.x = (int)ceil(hp.CellRow / (float)threadsPerBlock.x);
  numBlocks.y = (int)ceil(hp.CellCol / (float)threadsPerBlock.y);

  Cell_kernel_v2<<<numBlocks, threadsPerBlock, 0, stream[0]>>>(GPU_CellHistogram, Orientation, Gradient, hp);
 } else {
  threadsPerBlock = dim3(BOX_SIZE, BOX_SIZE);
  numBlocks = dim3((int)ceil(hp.CellRow / (float)threadsPerBlock.x), (int)ceil(hp.CellCol / (float)threadsPerBlock.y));
  Cell_kernel_v0<<<numBlocks, threadsPerBlock, 0, stream[0]>>>(GPU_CellHistogram, Orientation, Gradient, hp);
 }

 //-------------------------------------------------------------display_kernel-------------------------------------------------------------------------
  // display_kernel
 checkCuda(hipDeviceSynchronize());
 checkCuda(hipMemcpy(CPU_Hist,Displayhistogram , dp.TotalCells *4*4, hipMemcpyDeviceToHost));

//  $$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$ hipFree(TempDisplayhistogram);
//  $$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$ hipFree(Orientation); hipFree(Gradient);
 checkCuda(hipMalloc((void**)&GPU_odata, dp.DisplayImgSize));
 hipMemset(GPU_odata, 0, dp.DisplayImgSize);
 threadsPerBlock = dim3(4, 4, 4);
 numBlocks = dim3((int)ceil(dp.CellRow / (float)threadsPerBlock.x), (int)ceil(dp.CellCol / (float)threadsPerBlock.y));
//  //printf("\n\n...%d %d...\n\n",numBlocks.x,numBlocks.y); 

 if(display_kernel_v==1){
  display_kernel_v1<<<numBlocks, threadsPerBlock,0,stream[1]>>>(Displayhistogram,GPU_odata,dp);
 } else if(display_kernel_v==2){
  display_kernel_v2<<<numBlocks, threadsPerBlock,0,stream[1]>>>(Displayhistogram,GPU_odata,dp);
 } else {
  display_kernel_v0<<<numBlocks, threadsPerBlock,0,stream[1]>>>(Displayhistogram,GPU_odata,dp);
 }

 //-------------------------------------------------------------Block_kernel-------------------------------------------------------------------------
 // Block_kernel
 checkCuda(hipHostMalloc ((void**)&GPU_BlockHistogram,hp.TotalBlocks*sizeof(float)*hp.FeatureSize));
 threadsPerBlock = dim3(BOX_SIZE, BOX_SIZE);
 numBlocks = dim3((int)ceil(hp.BlockRow / (float)threadsPerBlock.x), (int)ceil(hp.BlockCol / (float)threadsPerBlock.y));
 //printf("\n\n...%d %d...\n\n",numBlocks.x,numBlocks.y); 

 if(Block_kernel_v==1){
  Block_kernel_v1<<<numBlocks, threadsPerBlock,0,stream[0]>>>(GPU_BlockHistogram, GPU_CellHistogram, hp);
 } else if(Block_kernel_v==2){
  Block_kernel_v2<<<numBlocks, threadsPerBlock,0,stream[0]>>>(GPU_BlockHistogram, GPU_CellHistogram, hp);
 } else if(Block_kernel_v==3){
  Block_kernel_v3<<<numBlocks, threadsPerBlock,0,stream[0]>>>(GPU_BlockHistogram, GPU_CellHistogram, hp);
 } else {
  Block_kernel_v0<<<numBlocks, threadsPerBlock,0,stream[0]>>>(GPU_BlockHistogram, GPU_CellHistogram, hp);
 }

//  //-------------------------------------------------------------Timings-------------------------------------------------------------------------
 hipEventRecord(time3, 0);
 
 checkCuda(hipMemcpyAsync(CPU_OutputArray, GPU_odata, dp.DisplayImgSize, hipMemcpyDeviceToHost,stream[1]));
 checkCuda(hipDeviceSynchronize());
 
//  //checkCuda(hipMemcpy(CPU_CellArray,GPU_CellHistogram , hp.TotalCells*sizeof(float)*hp.NumBins, hipMemcpyDeviceToHost));

 checkCuda(hipMemcpy(CPU_FeatureArray,GPU_BlockHistogram , hp.TotalBlocks*sizeof(float)*hp.FeatureSize, hipMemcpyDeviceToHost));
 
 	hipEventRecord(time4, 0);
	hipEventSynchronize(time1);
	hipEventSynchronize(time2);
	hipEventSynchronize(time3);
	hipEventSynchronize(time4);

	float totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime;

	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);

	Runtimes[0] = totalTime;
	Runtimes[1] = tfrCPUtoGPU;
	Runtimes[2] = kernelExecutionTime;
	Runtimes[3] = tfrGPUtoCPU;

 	Error:
  for(i=0;i<2;i++) hipStreamDestroy(stream[i]);
	hipFree(GPU_odata);
	hipFree(GPU_idata);
  hipFree(Orientation);
  hipFree(Gradient);
  hipFree(DisplayOrientation);
  // hipFree(GPU_BlockHistogram);
	hipFree(Displayhistogram);
  hipFree(TempDisplayhistogram);
 	hipEventDestroy(time1);
	hipEventDestroy(time2);
	hipEventDestroy(time3);
	hipEventDestroy(time4);
 
  return cudaStatus;
}